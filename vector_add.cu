
#include <hip/hip_runtime.h>
#include "stdio.h"
#include <iostream>

#define N 10000000

__global__ void vector_add(float *out, float *a, float *b, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    	out[idx] = a[idx] + b[idx];
}
__host__ void get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}
int main(){
    float *a, *b, *out;
    float *d_a, *d_b, *d_out;

    a = (float*)malloc(sizeof(float) * N);
    b = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);
    for (int i = 0; i < N; i++) {
	    *(a + i) = 1.0;
	    *(b + i) = 1.0;
    }

    // Allocate device memory for a
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) *N);
    hipMalloc((void**)&d_out, sizeof(float) *N);

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);
    dim3 gridDim(ceil(1.0*N/1024), 1, 1);
    dim3 blockDim(1024, 1, 1);
    vector_add<<<gridDim,blockDim>>>(d_out, d_a, d_b, N);
    hipDeviceSynchronize();
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);
    for (int i = 0; i < 10; i++)
	    printf("%f ", *(out + i));
   
    get_device_properties();
    // Cleanup after kernel execution
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    free(a);
    free(b);
    free(out);

    return 1;
}
